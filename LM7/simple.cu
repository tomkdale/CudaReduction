//CUDA reduction algorithm. simple approach
//Tom Dale
//11-20-18



#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
using namespace std;
#define N 100//number of input values
#define R 20//reduction factor
#define F (1+((N-1)/R))//how many values will be in the final output


//basicRun will F number of threads go through R number of values and put the average in z[tid]
__global__ void basicRun(double *a,double *z){
    int tid = blockDim.x*blockIdx.x + threadIdx.x;
    if(tid > F) return;
    double avg=0;
    for(int i= 0;i<R;i++){//get sum of input values in this threads domain
        avg += a[i+tid*R];
    }
    z[tid]=avg/R;//divide sum by total number of input values to get average
}




int main(){ 
    int bufferedSize = N + (N%R);//buffered size is closest evenly divisible by R value that is equal or greater than n
    double *a,*z;
    a = (double*)malloc(sizeof(double)*N);
    z = (double*)malloc(sizeof(double)*F);
    for(int i =0;i< N;i++){//set a to random values
        a[i]= rand() % 10;
        //a[i] = i;
    }

    for(int i = 0;i<(N%R);i++){//wrap around buffer. a will be extended to be evenly split by R.
        a[N+i] =a[i];//added buffer values will be equal to first few variables in the array as stated in problem
    }

    // for(int i =0;i< bufferedSize;i++){//print values to screen
    //     cout << a[i] << " ";
    // }
    // cout << endl;
    
    double *dev_a,*dev_z;//create device side variables
    hipMalloc((void**)&dev_a,sizeof(double)*bufferedSize);
    hipMalloc((void**)&dev_z,sizeof(double)*F);

    hipMemcpy(dev_a,a,sizeof(double)*bufferedSize,hipMemcpyHostToDevice);


    int gridSize =100;//number of blocks per grid remeber, should be 1 dimension
    int blockSize = 1024 ;//number of threads per block
    basicRun<<<gridSize,blockSize>>>(dev_a,dev_z);

    hipMemcpy(z,dev_z,sizeof(double)*F,hipMemcpyDeviceToHost);


    for(int i =0;i< F;i++){//output final reduced values
        cout << z[i] << " ";
    }
    

    hipFree(dev_a);
    hipFree(dev_z);


}